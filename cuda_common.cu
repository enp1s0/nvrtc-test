
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void cudaHandleError( hipError_t err,const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "CUDA Error\n%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
