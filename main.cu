#include "hip/hip_runtime.h"
/*
 * NVIDIA Runtime Compilation
 * 2017.11.22
 */

#include <iostream>
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include "cuda_common.h"

const int NUM_BLOCKS = 16;
const int NUM_THREADX = 16;

const char *kernel_code = "															\n\
						   extern \"C\"												\n\
						   __global__ void kernel(float *y,float *x,int max_tid){	\n\
							   int tid = blockIdx.x * blockDim.x + threadIdx.x;		\n\
							   if(tid >= max_tid)return;							\n\
							   y[tid] = __sinf(x[tid]);								\n\
						   }														\n";

int main(){
	hiprtcProgram program;
	hiprtcCreateProgram(&program,
			kernel_code,
			"kernel.cu",
			0,
			NULL,
			NULL);
	const char *options[] = {
		"--gpu-architecture=compute_60",
	};
	hiprtcResult result = hiprtcCompileProgram(program,
			1,
			options);
	size_t log_size;
	hiprtcGetProgramLogSize(program,&log_size);
	char *log = new char[log_size];
	hiprtcGetProgramLog(program,log);
	std::cout<<log<<std::endl;
	delete [] log;
	if(result != HIPRTC_SUCCESS){
		std::cerr<<"Compilation failed"<<std::endl;
		return 1;
	}
	size_t ptx_size;
	hiprtcGetCodeSize(program,&ptx_size);
	char *ptx = new char [ptx_size];
	hiprtcGetCode(program,ptx);
	hiprtcDestroyProgram(&program);

	hipDevice_t cuDevice;
	hipCtx_t cuContext;
	hipModule_t cuModule;
	hipFunction_t cuFunction;
	hipInit(0);
	hipDeviceGet(&cuDevice,0);
	hipCtxCreate(&cuContext,0,cuDevice);
	hipModuleLoadDataEx(&cuModule,ptx,0,0,0);
	hipModuleGetFunction(&cuFunction,cuModule,"kernel");

	size_t n = NUM_BLOCKS * NUM_THREADX;
	size_t mem_size = sizeof(float) * n;
	float *dx,*dy;
	float *hx,*hy;

	CUDA_HANDLE_ERROR( hipMalloc((void**)&dx,mem_size) );
	CUDA_HANDLE_ERROR( hipMalloc((void**)&dy,mem_size) );
	CUDA_HANDLE_ERROR( hipHostMalloc((void**)&hx,mem_size) );
	CUDA_HANDLE_ERROR( hipHostMalloc((void**)&hy,mem_size) );
	for(int i = 0;i < n;i++)hx[i] = i* 3.141592f/n;
	CUDA_HANDLE_ERROR( hipMemcpy(dx,hx,mem_size,hipMemcpyHostToDevice) );

	void *args[] = {&dy,&dx,&n};
	hipModuleLaunchKernel(cuFunction,
				NUM_BLOCKS,1,1,
				NUM_THREADX,1,1,
				0,NULL,
				args,0);
	hipCtxSynchronize();
	
	CUDA_HANDLE_ERROR( hipMemcpy(hy,dy,mem_size,hipMemcpyDeviceToHost) );
	for(int i = 0;i < n;i++){
		std::cout<<"sin( "<<hx[i]<<" ) = "<<hy[i]<<" ;";
		if((i+1)%6 == 0)std::cout<<std::endl;
	}

	CUDA_HANDLE_ERROR( hipFree(dx) );
	CUDA_HANDLE_ERROR( hipFree(dy) );
	CUDA_HANDLE_ERROR( hipHostFree(hx) );
	CUDA_HANDLE_ERROR( hipHostFree(hy) );

	delete [] ptx;
	hipModuleUnload(cuModule);
	hipCtxDestroy(cuContext);
}
